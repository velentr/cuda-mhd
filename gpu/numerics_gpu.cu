#include "hip/hip_runtime.h"
/* Author: Yubo Su
 * Numerics suite for cuda-mhd CPU implementation
 */
#include <stdlib.h>
#include <string.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

#include "numerics_gpu.h"

__global__ void cuda_update(double* yn, int leny)
{
    /* Update computations of P^*, EPmDV */
    int n = (int)cbrt((double)(leny / NUM_COMPS));
    int i, j, k;
    float bdotv; /* useul for precomputation */
    for(i = 0; i < n; i++)
    {
        for(j = 0; j < n; j++)
        {
            for(k = 0; k < n; k++)
            {
                /* store this since used thrice */
                bdotv = yn[U2(n, i, j, k)] * yn[U5(n, i, j, k)] +
                    yn[U3(n, i, j, k)] * yn[U6(n, i, j, k)] +
                    yn[U4(n, i, j, k)] * yn[U7(n, i, j, k)];

                /* compute P^*, EPmDV */
                yn[U9(n, i, j, k)] = (GAMMA - 1) *
                    (yn[U8(n, i, j, k)] -
                        (yn[U2(n, i, j, k)] * yn[U2(n, i, j, k)] +
                         yn[U3(n, i, j, k)] * yn[U3(n, i, j, k)] +
                         yn[U4(n, i, j, k)] * yn[U4(n, i, j, k)]) /
                        (2 * yn[U1(n, i, j, k)]) +
                        (yn[U5(n, i, j, k)] * yn[U5(n, i, j, k)] +
                         yn[U6(n, i, j, k)] * yn[U6(n, i, j, k)] +
                         yn[U7(n, i, j, k)] * yn[U7(n, i, j, k)]) / 2);

                yn[U10(n, i, j, k)] = - (yn[U8(n, i, j, k)] + yn[U9(n, i, j, k)])
                    * yn[U2(n, i, j, k)] + yn[U5(n, i, j, k)] * bdotv;
                yn[U11(n, i, j, k)] = - (yn[U8(n, i, j, k)] + yn[U9(n, i, j, k)])
                    * yn[U3(n, i, j, k)] + yn[U6(n, i, j, k)] * bdotv;
                yn[U12(n, i, j, k)] = - (yn[U8(n, i, j, k)] + yn[U9(n, i, j, k)])
                    * yn[U4(n, i, j, k)] + yn[U7(n, i, j, k)] * bdotv;
            }
        }
    }
}


__global__ void cuda_step(void (*f)(double *, double *, int), double dt,
        double *yn, double *ynew, double *dy, int leny, double *scratch,
        double *scratch2)
/* Given some dy/dt = f(y), compute the dy for a given yn and store into yn +
 * dy into ynew. Uses Runge-Kutta:
 * k1 = f(y), k2 = f(y + k1 * dt/2), k3 = f(y + k2 * dt/2),
 *      k4 = f(y + k3 * dt), ynew = yn + (k1 + 2k2 + 2k3 + k4)/6 * dt
 *
 * 2 scratch vectors required to not nuke in case ynew = yn (overwrite)
 *
 * Input:
 *  void (*f)           : computes dy/dt = f(y), stores into second double*
 *  double dt            : timestep dt
 *  double* yn           : current y
 *  double* ynew         : new y
 *  double* dy           : get return from (*f) (malloc in caller)
 *  int leny            : length of y vector
 *  double* scratch[2]   : scratch vector, simply to avoid re-mallocing,
 *                          also leny
 */
{
    /* organization scheme:
     * yn = y
     * scratch2 = k1 + ... (running tally)
     * scratch = y + ...
     * dy = f(y + ...)
     *
     * then copy correct value into ynew */
    int i;

    (*f)(yn, dy, leny); /* compute k1 into dy */
    /* scratch = yn + k1 * dt/2 */
    for (i = 0; i < leny; i++)
    {
        scratch[i] = yn[i] + dy[i] * dt / 2; /* y + k1 * dt/2 */
        scratch2[i] = dy[i] / 6; /* k1 / 6 */
    }

    (*f)(scratch, dy, leny); /* compute k2 into dy */
    for (i = 0; i < leny; i++)
    {
        scratch[i] = yn[i] + dy[i] * dt / 2; /* y + k2 * dt/2 */
        scratch2[i] += dy[i] / 3; /* (k1 + 2k2) / 6 */
    }

    (*f)(scratch, dy, leny); /* compute k3 into dy */
    for (i = 0; i < leny; i++)
    {
        scratch[i] = yn[i] + dy[i] * dt; /* y + k3 * dt */
        scratch2[i] += dy[i] / 3; /* (k1 + 2k2 + 2k3) / 6 */
    }

    (*f)(scratch, dy, leny); /* compute k4 into dy */
    for (i = 0; i < leny; i++)
    {
        /* dt * (k1 + 2k2 + 2k3 + k4) / 6 + yn */
        ynew[i] = dt * (scratch2[i] + dy[i] / 6) + yn[i];
    }
}

double* solve(void (*f)(double*, double*, int),
        double* y0, double dt, int nsteps, int leny, int save_skip,
        size_t nblk, size_t thr)
/* Given some initial y(t = 0) and dy/dt = f(y) (autonomous ODE), computes
 * y(nsteps * dt) by iterating y_n -> y_{n+1} via step. y assumed in
 * generality to be a vector
 *
 * Input:
 *  void (*f)           : computes dy/dt = f(y), stores into second double*
 *  double* init         : y(t=0)
 *  double dt            : dt timestep
 *  int nsteps          : number of timesteps to evolve
 *  int leny            : length of y vector
 *  int save_skip       : every save_skip timesteps, save trajectory
                        : use only divisible by nsteps please
 * Output:
 *  double*              : (nsteps + 1) * leny vector containing y(t), contains
 *                        timesteps dt * [0,nsteps / saveskip]
 */
{
    /* stores total trajectory */
    double *dy;
    double *scratch;
    double *scratch2;
    double *ret_dev;
    double *ret = (double *) malloc((nsteps / save_skip + 1) * leny * sizeof(double));
    double *source, * dest; /* temporaries */
    int i, j;

    /* Allocate memory for working set. */
    hipMalloc((void **) &dy,        leny * sizeof(double));
    hipMalloc((void **) &scratch,   leny * sizeof(double));
    hipMalloc((void **) &scratch2,  leny * sizeof(double));

    /* Memory for output buffer. */
    hipMalloc((void **)&ret_dev,
            (nsteps / save_skip + 1) * leny * sizeof(double));

    /* start with ret = y0 */
    hipMemcpy(ret_dev, y0, leny * sizeof(double), hipMemcpyDeviceToDevice);

    /* Run an initial update to get the correct P* and temp values. */
    cuda_update<<<nblk, thr>>>(y0, leny);

    /* at each step, get ynew into ret, else edit in place */
    for (i = 0; i < nsteps / save_skip; i++)
    {
        source = ret_dev + i * leny;
        dest = source + leny; /* save into here */

        for (j = 0; j < save_skip; j++)
        {
            /* Execute a single step in the simulation. */
            cuda_step<<<nblk, thr>>>(f, dt, source, dest, dy, leny, scratch,
                    scratch2);

            /* Update the dataset. */
            cuda_update<<<nblk, thr>>>(dest, leny);

            /* first iteration of this loop doesn't overwrite source */
            source = dest;
        }
    }

    /* Copy return buffer back to host. */
    hipMemcpy(ret, ret_dev, (nsteps / save_skip + 1) * leny * sizeof(double),
            hipMemcpyDeviceToHost);

    /* cleanup */
    hipFree(dy);
    hipFree(scratch);
    hipFree(scratch2);

    return ret;
}
